#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

__global__ static void kernel_tc(uint64_t *__restrict__ triangleCounts, //!< per-edge triangle counts
                                 const uint32_t *const edgeSrc,         //!< node ids for edge srcs
                                 const uint32_t *const edgeDst,         //!< node ids for edge dsts
                                 const uint32_t *const rowPtr,          //!< source node offsets in edgeDst
                                 const size_t numEdges                  //!< how many edges to count triangles for
) {
  // Determine the source and destination node for the edge
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  
  if(idx >= numEdges) 
    return;

  int src = edgeSrc[idx];
  int dst = edgeDst[idx];

  // Use the row pointer array to determine the start and end of the neighbor list in the column index array
  int u = rowPtr[src];
  int v = rowPtr[dst];
  int uEnd = rowPtr[src + 1];
  int vEnd = rowPtr[dst + 1];

  // Determine how many elements of those two arrays are common
  int w1 = edgeDst[u];
  int w2 = edgeDst[v];
  uint64_t ans = 0;
  while(u < uEnd && v < vEnd)
  {
    if(w1 > w2)
    {
      w2 = edgeDst[++ v];
    }
    else if(w1 < w2)
    {
      w1 = edgeDst[++ u];
    }
    else
    {
      w1 = edgeDst[++ u];
      w2 = edgeDst[++ v];
      ans ++;
    }
  }
  triangleCounts[idx] = ans;
}

__global__ void kernel_tc_bs()
{
}

uint64_t count_triangles(const pangolin::COOView<uint32_t> view, const int mode) {
  //@@ create a pangolin::Vector (uint64_t) to hold per-edge triangle counts
  // Pangolin is backed by CUDA so you do not need to explicitly copy data between host and device.
  // You may find pangolin::Vector::data() function useful to get a pointer for your kernel to use.
  pangolin::Vector<uint64_t> pgl(view.nnz(), 0);


  dim3 dimBlock(512);
  //@@ calculate the number of blocks needed
  dim3 dimGrid (ceil(1.0 * view.nnz() / dimBlock.x));

  if (mode == 1) 
  {
    //@@ launch the linear search kernel here
    kernel_tc<<<dimGrid, dimBlock>>>(pgl.data(), view.row_ind(), view.col_ind(), view.row_ptr(), view.nnz());
    hipDeviceSynchronize();

  } 
  else if (2 == mode) 
  {
    //@@ launch the hybrid search kernel here
    // your_kernel_name_goes_here<<<dimGrid, dimBlock>>>(...)

  }
  else 
  {
    assert("Unexpected mode");
    return uint64_t(-1);
  }

  //@@ do a global reduction (on CPU or GPU) to produce the final triangle count
  uint64_t total = 0;
  for(int i = 0 ; i < view.nnz() ; i ++)
  {
    total += pgl[i];
  }
  return total;
}
