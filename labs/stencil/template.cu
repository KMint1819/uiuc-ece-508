#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "helper.hpp"

#define TILE_SIZE 30

__global__ void kernel(int *A0, int *Anext, int nx, int ny, int nz) {
  #define in(i, j, k) A0[((k)*ny + (j))*nx + (i)]
  #define out(i, j, k) Anext[((k)*ny + (j))*nx + (i)]

  // INSERT CODE HERE
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int i = bx * TILE_SIZE + tx;
  int j = by * TILE_SIZE + ty;

  if(i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1)
  {
    int prev = 0;
    int cur = in(i, j, 0);
    int next = in(i, j, 1);

    for(int k = 1 ; k < nz - 1 ; k ++)
    { 
      prev = cur;
      cur = next;
      next = in(i, j, k + 1);

      out(i, j, k) = -6 * cur 
                      + prev 
                      + next
                      + in(i - 1, j, k) 
                      + in(i, j - 1, k) 
                      + in(i + 1, j, k) 
                      + in(i, j + 1, k);
    }
  }
  #undef in
  #undef out 
}

void launchStencil(int* A0, int* Anext, int nx, int ny, int nz) {
  dim3 dimGrid(ceil(1.0 * nx / TILE_SIZE), ceil(1.0 * ny / TILE_SIZE), 1);
  dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

  kernel<<<dimGrid, dimBlock>>>(A0, Anext, nx, ny, nz);
}


static int eval(const int nx, const int ny, const int nz) {

  // Generate model
  const auto conf_info = std::string("stencil[") + std::to_string(nx) + "," + 
                                                   std::to_string(ny) + "," + 
                                                   std::to_string(nz) + "]";
  INFO("Running "  << conf_info);

  // generate input data
  timer_start("Generating test data");
  std::vector<int> hostA0(nx * ny * nz);
  generate_data(hostA0.data(), nx, ny, nz);
  std::vector<int> hostAnext(nx * ny * nz);

  timer_start("Allocating GPU memory.");
  int *deviceA0 = nullptr, *deviceAnext = nullptr;
  CUDA_RUNTIME(hipMalloc((void **)&deviceA0, nx * ny * nz * sizeof(int)));
  CUDA_RUNTIME(hipMalloc((void **)&deviceAnext, nx * ny * nz * sizeof(int)));
  timer_stop();

  timer_start("Copying inputs to the GPU.");
  CUDA_RUNTIME(hipMemcpy(deviceA0, hostA0.data(), nx * ny * nz * sizeof(int), hipMemcpyDefault));
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  timer_start("Performing GPU convlayer");
  launchStencil(deviceA0, deviceAnext, nx, ny, nz);
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  timer_start("Copying output to the CPU");
  CUDA_RUNTIME(hipMemcpy(hostAnext.data(), deviceAnext, nx * ny * nz * sizeof(int), hipMemcpyDefault));
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  // verify with provided implementation
  timer_start("Verifying results");
  verify(hostAnext.data(), hostA0.data(), nx, ny, nz);
  timer_stop();

  CUDA_RUNTIME(hipFree(deviceA0));
  CUDA_RUNTIME(hipFree(deviceAnext));

  return 0;
}



TEST_CASE("Stencil", "[stencil]") {

  SECTION("[dims:32,32,32]") {
    eval(32,32,32);
  }
  SECTION("[dims:30,30,30]") {
    eval(30,30,30);
  }
  SECTION("[dims:29,29,29]") {
    eval(29,29,29);
  }
  SECTION("[dims:31,31,31]") {
    eval(31,31,31);
  }
  SECTION("[dims:29,29,2]") {
    eval(29,29,29);
  }
  SECTION("[dims:1,1,2]") {
    eval(1,1,2);
  }
  SECTION("[dims:512,512,64]") {
    eval(512,512,64);
  }

}
